
#include <hip/hip_runtime.h>
extern "C"
__device__
int rt_add(int x, int y) {
    return x + y;
}

extern "C"
__device__
void *_rt_alloc_array(int elemSize, int length) {
    int buffSz = elemSize * length + 8;
    char *buff = (char *)malloc(buffSz);
    memset(buff, 0, buffSz);

    *((int *)&buff[0]) = length;
    *((int *)&buff[4]) = elemSize;

    return (void *)&buff[8];
}
