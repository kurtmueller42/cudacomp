
#include <hip/hip_runtime.h>
extern "C"
__device__
int rt_add(int x, int y) {
    return x + y;
}

extern "C"
__device__
int *rt_calloc(int count, int size) {
    int *buff = (int *)malloc(count * size);
    memset(buff, 0, count * size);
    return buff;
}
